//
// Created by deano on 27/04/16.
//

#include "biasedconvolutionnnlayer.h"

BiasedConvolutionNNLayer::BiasedConvolutionNNLayer(
        CudaContext::ptr _context,
        int _inputWidth, int _inputHeight, int _inputChannels,
        int _outputChannels, int _kernelSize, int _batchSize ) :
        ctx(_context),
        inputWidth(_inputWidth),
        inputHeight(_inputHeight),
        inputChannels(_inputChannels),
        outputChannels(_outputChannels),
        kernelSize(_kernelSize),
        batchSize(_batchSize),
        outputWidth((inputWidth - kernelSize) + 1),
        outputHeight((inputHeight - kernelSize) + 1),
        weights(inputChannels * kernelSize * kernelSize * outputChannels),
        bias(outputChannels) {

    checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          batchSize,
                                          inputChannels, inputHeight, inputWidth));

    checkCUDNN(hipdnnCreateFilterDescriptor(&filterDescriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filterDescriptor,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          HIPDNN_TENSOR_NCHW,
                                          outputChannels,
                                          inputChannels,
                                          kernelSize,
                                          kernelSize));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&forwardConvolutionDescriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(forwardConvolutionDescriptor,
                                               0, 0, // padding
                                               1, 1, // filter stride
                                               1, 1, // scaling
                                               HIPDNN_CROSS_CORRELATION));

    int n, c, h, w;
    // Find dimension of convolution output
    // get the output to account for scaling/stride/padding etc.
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(forwardConvolutionDescriptor,
                                                     inputTensorDescriptor,
                                                     filterDescriptor,
                                                     &n, &c, &h, &w));

    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          n, c,
                                          h, w));

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(ctx->getCudnnHandle(),
                                                   inputTensorDescriptor,
                                                   filterDescriptor,
                                                   forwardConvolutionDescriptor,
                                                   outputTensorDescriptor,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   0,
                                                   &forwardConvolutionAlgorithm));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(ctx->getCudnnHandle(),
                                                       inputTensorDescriptor,
                                                       filterDescriptor,
                                                       forwardConvolutionDescriptor,
                                                       outputTensorDescriptor,
                                                       forwardConvolutionAlgorithm,
                                                       &forwardWorkspaceSize));

    ctx->reserveWorkspace(forwardWorkspaceSize);

    checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          1, outputChannels,
                                          1, 1));
}

BiasedConvolutionNNLayer::~BiasedConvolutionNNLayer() {

    checkCUDNN(hipdnnDestroyConvolutionDescriptor(forwardConvolutionDescriptor));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterDescriptor));

    checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensorDescriptor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensorDescriptor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDescriptor));

}

void BiasedConvolutionNNLayer::forwardPropogate( const float alpha, const float beta, const float *x, float *y ) {

    void *const workspace = ctx->grabWorkspace(forwardWorkspaceSize);

    checkCUDNN(hipdnnConvolutionForward(ctx->getCudnnHandle(),
                                       &alpha,
                                       inputTensorDescriptor,
                                       x,
                                       filterDescriptor,
                                       weights.data(),
                                       forwardConvolutionDescriptor,
                                       forwardConvolutionAlgorithm,
                                       workspace,
                                       forwardWorkspaceSize,
                                       &beta,
                                       outputTensorDescriptor,
                                       y));

    ctx->releaseWorkspace(workspace, forwardWorkspaceSize);

}

void BiasedConvolutionNNLayer::backPropogate( const float alpha, const float beta, const float *dy, float *db ) {
    checkCudaErrors(hipdnnConvolutionBackwardBias(ctx->getCudnnHandle(),
                                                 &alpha,
                                                 outputTensorDescriptor,
                                                 dy,
                                                 &beta,
                                                 biasTensorDescriptor,
                                                 db));
}