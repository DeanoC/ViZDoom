//
// Created by deano on 27/04/16.
//

#include <hipDNN.h>
#include "cudainit.h"
#include "cudacontext.h"
#include "biasedconvolutionnnlayer.h"

#if USE_HALF_FLOATS
#define CUDNN_DATA_HALF_OR_FLOAT HIPDNN_DATA_HALF
#else
#define CUDNN_DATA_HALF_OR_FLOAT HIPDNN_DATA_FLOAT
#endif

BiasedConvolutionNNLayer::BiasedConvolutionNNLayer(
        CudaContext::ptr _context,
        int _inputWidth, int _inputHeight, int _inputChannels,
        int _outputChannels, int _kernelSize, int _batchSize ) :
        ctx(_context),
        inputWidth(_inputWidth),
        inputHeight(_inputHeight),
        inputChannels(_inputChannels),
        outputChannels(_outputChannels),
        kernelSize(_kernelSize),
        batchSize(_batchSize),
        outputWidth((inputWidth - kernelSize) + 1),
        outputHeight((inputHeight - kernelSize) + 1),
        weights(inputChannels * kernelSize * kernelSize * outputChannels),
        bias(outputChannels) {

    checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          batchSize,
                                          inputChannels, inputHeight, inputWidth));

    checkCUDNN(hipdnnCreateFilterDescriptor(&filterDescriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filterDescriptor,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          HIPDNN_TENSOR_NCHW,
                                          outputChannels,
                                          inputChannels,
                                          kernelSize,
                                          kernelSize));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&forwardConvolutionDescriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(forwardConvolutionDescriptor,
                                               0, 0, // padding
                                               1, 1, // filter stride
                                               1, 1, // scaling
                                               HIPDNN_CROSS_CORRELATION));

    int n, c, h, w;
    // Find dimension of convolution output
    // get the output to account for scaling/stride/padding etc.
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(forwardConvolutionDescriptor,
                                                     inputTensorDescriptor,
                                                     filterDescriptor,
                                                     &n, &c, &h, &w));

    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          n, c,
                                          h, w));

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(ctx->getCudnnHandle(),
                                                   inputTensorDescriptor,
                                                   filterDescriptor,
                                                   forwardConvolutionDescriptor,
                                                   outputTensorDescriptor,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   0,
                                                   &forwardConvolutionAlgorithm));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(ctx->getCudnnHandle(),
                                                       inputTensorDescriptor,
                                                       filterDescriptor,
                                                       forwardConvolutionDescriptor,
                                                       outputTensorDescriptor,
                                                       forwardConvolutionAlgorithm,
                                                       &forwardWorkspaceSize));

    checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          1, outputChannels,
                                          1, 1));
}

BiasedConvolutionNNLayer::~BiasedConvolutionNNLayer() {

    checkCUDNN(hipdnnDestroyConvolutionDescriptor(forwardConvolutionDescriptor));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterDescriptor));

    checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensorDescriptor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensorDescriptor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDescriptor));

}