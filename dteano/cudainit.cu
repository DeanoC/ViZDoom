//
// Created by deano on 26/04/16.
//

#include "cudainit.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>

#include <algorithm>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <sstream>
#include <vector>

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

int cudaInit() {
    int num_gpus;
    checkCudaErrors(hipGetDeviceCount(&num_gpus));

    return num_gpus;
}