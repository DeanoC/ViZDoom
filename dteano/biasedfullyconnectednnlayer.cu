//
// Created by deano on 27/04/16.
//

#include "biasedfullyconnectednnlayer.h"

BiasedFullyConnectedNNLayer::BiasedFullyConnectedNNLayer( int _inputSize, int _outputSize, int _batchSize ) :
        inputSize(_inputSize),
        outputSize(_outputSize),
        batchSize(_batchSize),
        weights(inputSize * outputSize),
        bias(outputSize) {
    checkCUDNN(hipdnnCreateTensorDescriptor(&tensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_FLOAT_OR_HALF,
                                          batchSize,
                                          outputSize, 1, 1));

    checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          1, outputChannels,
                                          1, 1));
}

BiasedFullyConnectedNNLayer::~BiasedFullyConnectedNNLayer() {

    checkCUDNN(hipdnnDestroyTensorDescriptor(tensorDescriptor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDescriptor));
}