//
// Created by deano on 27/04/16.
//

#include "biasedfullyconnectednnlayer.h"

BiasedFullyConnectedNNLayer::BiasedFullyConnectedNNLayer( int _inputSize, int _outputSize, int _batchSize ) :
        inputSize(_inputSize),
        outputSize(_outputSize),
        batchSize(_batchSize),
        weights(inputSize * outputSize),
        bias(outputSize) {

    checkCUDNN(hipdnnCreateTensorDescriptor(&tensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          batchSize,
                                          outputSize, 1, 1));

    checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DATA_HALF_OR_FLOAT,
                                          1, outputSize,
                                          1, 1));
}

BiasedFullyConnectedNNLayer::~BiasedFullyConnectedNNLayer() {

    checkCUDNN(hipdnnDestroyTensorDescriptor(tensorDescriptor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDescriptor));
}